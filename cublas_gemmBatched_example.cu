#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    int m = 2;
    int n = 2;
    int k = 2;
    int lda = 2;
    int ldb = 2;
    int ldc = 2;
    int batch_count = 2;

    /*
     *   A = | 1.0 | 2.0 | 5.0 | 6.0 |
     *       | 3.0 | 4.0 | 7.0 | 8.0 |
     *
     *   B = | 5.0 | 6.0 |  9.0 | 10.0 |
     *       | 7.0 | 8.0 | 11.0 | 12.0 |
     */

    std::vector<std::vector<data_type>> A_array;
    std::vector<std::vector<data_type>> B_array;
    std::vector<std::vector<data_type>> C_array(batch_count, std::vector<data_type>(m * n));

    const data_type alpha = 1.0;
    const data_type beta = 0.0;

    data_type **d_A_array = nullptr;
    data_type **d_B_array = nullptr;
    data_type **d_C_array = nullptr;

    std::vector<data_type *> d_A(batch_count, nullptr);
    std::vector<data_type *> d_B(batch_count, nullptr);
    std::vector<data_type *> d_C(batch_count, nullptr);

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    for(int i=0;i<batch_count;i++){
        std::vector<data_type> tmpA;
        gen_random_matrix_2_d<data_type>(&tmpA,m,n,&lda,&d_A[i]);
        std::vector<data_type> tmpB;
        gen_random_matrix_2_d<data_type>(&tmpB,n,k,&ldb,&d_B[i]);
        A_array.push_back(tmpA);
        B_array.push_back(tmpB);
        CUDA_CHECK(
            hipMalloc(reinterpret_cast<void **>(&d_C[i]), sizeof(data_type) * C_array[i].size()));
    }
    /* step 2: copy data to device */

    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A_array), sizeof(data_type *) * batch_count));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B_array), sizeof(data_type *) * batch_count));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C_array), sizeof(data_type *) * batch_count));

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(data_type *) * batch_count,
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(data_type *) * batch_count,
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C_array, d_C.data(), sizeof(data_type *) * batch_count,
                               hipMemcpyHostToDevice, stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasDgemmBatched(cublasH, transa, transb, m, n, k, &alpha, d_A_array, lda,
                                    d_B_array, ldb, &beta, d_C_array, ldc, batch_count));

    /* step 4: copy data to host */
    for (int i = 0; i < batch_count; i++) {
        CUDA_CHECK(hipMemcpyAsync(C_array[i].data(), d_C[i], sizeof(data_type) * C_array[i].size(),
                                   hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   C = | 19.0 | 22.0 | 111.0 | 122.0 |
     *       | 43.0 | 50.0 | 151.0 | 166.0 |
     */

    printf("C[0]\n");
    print_matrix(m, n, C_array[0].data(), ldc);
    printf("=====\n");

    printf("C[1]\n");
    print_matrix(m, n, C_array[1].data(), ldc);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    CUDA_CHECK(hipFree(d_C_array));
    for (int i = 0; i < batch_count; i++) {
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
    }

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
